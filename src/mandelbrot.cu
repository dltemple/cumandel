#include <hip/hip_runtime.h>


__global__ void mandelbrotKernel(unsigned char *img, const int width, const int height, const float x_min, const float x_max, const float y_min, const float y_max, const int max_iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx < width && idy < height) {
        float x = x_min + (x_max - x_min) * idx / (width - 1);
        float y = y_min + (y_max - y_min) * idy / (height - 1);

        float real = x;
        float imag = y;
        int value = 0;

        for (int i = 0; i < max_iterations; i++) {
            float r2 = real * real;
            float i2 = imag * imag;

            if (r2 + i2 > 4.0f) {
                break;
            }

            imag = 2 * real * imag + y;
            real = r2 - i2 + x;

            value++;
        }

        img[index] = static_cast<unsigned char>((value % 256));
    }
}
